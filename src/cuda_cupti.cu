#include "hip/hip_runtime.h"
#include <iostream>

#include "cupti.h"

using std::cout;
using std::cerr;

// wrapper macro for CUDA API calls
#define CUDA_ERR_SAFE(cuda_call) { \
hipError_t cuda_error_code = (cuda_call); \
if (hipSuccess != cuda_error_code) { \
    cerr << "CUDA error " << cuda_error_code << ": '" << hipGetErrorString(cuda_error_code) << "'; line " << __LINE__ << " file " << __FILE__ << "; aborting\n"; \
    exit(EXIT_FAILURE); \
} \
}

// wrapper macro for CUPTI API calls
#define CUPTI_ERR_SAFE(cupti_call) { \
CUptiResult cupti_result = (cupti_call); \
if (cupti_result != CUPTI_SUCCESS) { \
    const char* cupti_err_str; \
    cuptiGetResultString(cupti_result, &cupti_err_str); \
    cerr << "CUPTI error " << cupti_result << ": '" << cupti_err_str << "'; line " << __LINE__ << " file " << __FILE__ << "; aborting\n"; \
    exit(EXIT_FAILURE); \
} \
}

// callback for CUPTI to request a buffer to populate with the requested activity records
void CUPTIAPI cuptiBufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords) {
    // 16 MB; docs recommend 1 to 10 MB
    //   this is a lot more than needed for this workload though
    *size = 16 * 1024 * 1024;
    *buffer = (uint8_t*) malloc(*size);
    // 0 means CUPTI will return all records (but still won't overflow buffer)
    *maxNumRecords = 0;
}

// callback for CUPTI to invoke when activity recording is complete
//  - enumerates and ouputs the activity records
void CUPTIAPI cuptiBufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize) {
    CUptiResult status;
    CUpti_Activity *record = NULL;

    cout << "CUPTI buffer completed\n";

    do {
        status = cuptiActivityGetNextRecord(buffer, validSize, &record);
        if (status == CUPTI_SUCCESS) {
            // Process record (kernel execution times, etc.)
            if (record->kind == CUPTI_ACTIVITY_KIND_KERNEL) {
                CUpti_ActivityKernel7 *kernel = (CUpti_ActivityKernel7 *)record;
                printf("Kernel execution: %s; grid size (%d,%d,%d); block size (%d,%d,%d); time %llu ns\n",
                    kernel->name,
                    kernel->gridX, kernel->gridY, kernel->gridZ,
                    kernel->blockX, kernel->blockY, kernel->blockZ,
                    (unsigned long long)(kernel->end - kernel->start));
                printf("                  registers per thread  %d\n", kernel->registersPerThread);
                printf("                  static shared memory  %d bytes\n", kernel->staticSharedMemory);
                printf("                  dynamic shared memory %d bytes\n", kernel->dynamicSharedMemory);
                printf("                  local memory per thread %u bytes\n", kernel->localMemoryPerThread);
            }
            else if (record->kind == CUPTI_ACTIVITY_KIND_SHARED_ACCESS) {
                CUpti_ActivitySharedAccess *sharedAccess = (CUpti_ActivitySharedAccess *)record;
                printf("Shared access: number of shared transactions %lu\n", sharedAccess->sharedTransactions);
            }
            else if (record->kind == CUPTI_ACTIVITY_KIND_FUNCTION) {
                CUpti_ActivityModule *module = (CUpti_ActivityModule *)record;
                printf("Function activity: module id %u\n", module->contextId);
            }
            else {
                printf("Record of type %d (not unpacking)\n", record->kind);
            }
        } else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
            break;
        } else {
            // here's a neat trick--the macro works just fine if fed the return value
            //   of a CUPTI function call instead of an actual CUPTI function call
            CUPTI_ERR_SAFE( status );
        }
    } while (1);
}

// trivial kernel to invoke to warm up a CUDA device; called before profiling
//   to ensure initialization activities/times don't affect profiling
__global__ void warm_up() {
    // this is not used as an index here, but this is how we would calculate the
    //   index into the data for a one-dimensional array
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    double x1 = 0.0;
    double x2 = 2.0;
    x1 = x1 + (double) index * x2 + (double) threadIdx.x;
}

// simple kernel to run the standard calculation on a single point
__global__ void calculate(double *points, size_t num_points) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_points) {
        points[index] *= static_cast<double>(index);
    }
}

// kernel to run the standard calculation on a single point but with stepping--all
//   points get calculated but the threads in a block will calculate points separated
//   by the step count instead of consecutive points (pass in a step count of 1 to
//   have the same behavior as the plain calculate kernel)
__global__ void calculate_with_step(double *points, size_t step_size, size_t num_points) {
    const unsigned int offset = blockIdx.x % step_size;
    const unsigned int span_num = (blockIdx.x / step_size) * step_size * blockDim.x;
    const unsigned int index =  span_num + threadIdx.x * step_size + offset;
    if (index < num_points) {
        points[index] *= static_cast<double>(index);
    }
}

// kernel to run the standard calculation on a single point but with stepping, and
//   copying the data to shared memory
//  - extra, meaningless calculation steps added to up the number of clock cycles
__global__ void calculate_with_step_shared_mem(double *points, size_t step_size, size_t num_points) {
    extern __shared__ double shared_data[];
    const unsigned int offset = blockIdx.x % step_size;
    const unsigned int span_num = (blockIdx.x / step_size) * step_size * blockDim.x;
    const unsigned int local_index = threadIdx.x * step_size + offset;
    const unsigned int global_index = span_num + local_index;
    if (global_index < num_points) {
        double original_value = points[global_index];
        // arbitrary calculations; hit the shared data multiple times
        shared_data[local_index] = original_value;
        shared_data[local_index] += original_value;
        shared_data[local_index] *= original_value;
        shared_data[local_index] -= original_value;
        shared_data[local_index] *= original_value;
        shared_data[local_index] /= original_value;
        shared_data[local_index] -= original_value;
        shared_data[local_index] /= original_value;
        shared_data[local_index] += original_value;
        shared_data[local_index] *= original_value;
        // final, 'correct' calculation
        shared_data[local_index] = original_value;
        shared_data[local_index] *= static_cast<double>(global_index);
        points[global_index] = shared_data[local_index];
    }
}

// Debug function to output the points data; used to diagnose mismatches
//   is expected and actual calculation outcomes
void data_dump(double *points, size_t num_points) {
    for (size_t index = 0; index < num_points; ++index) {
        if (0 == index % 10) {
            cout << "\n" << points[index];
        }
        else {
            cout << "\t" << points[index];
        }
    }
    cout << "\n\n";
}

// compare 'equality' of doubles for both absolute and relative tolerance
bool doubles_within_tolerance(const double x, const double y) {
    const double tolerance = 0.0000001;
    const double diff = abs(x - y);
    if (diff < tolerance) {
        return true;
    }
    const double maxd = (abs(x) > abs(y)) ? abs(x) : abs(y);
    return diff < tolerance * maxd;
}

// enumerate devices and output some useful properties
void devices_info(size_t total_data_bytes) {
    int cuda_dev_count;
    CUDA_ERR_SAFE( hipGetDeviceCount(&cuda_dev_count) );
    if (cuda_dev_count < 1) {
        cerr << "No CUDA devices available\n";
        exit(EXIT_FAILURE);
    }

        for (int dev_num = 0; dev_num < cuda_dev_count; ++dev_num) {
        hipDeviceProp_t cuda_dev_properties;
        CUDA_ERR_SAFE( hipGetDeviceProperties(&cuda_dev_properties, dev_num) );
        cout << "\ndevice number: " << dev_num << "\n";
        cout << "device name: " << cuda_dev_properties.name << "\n";
        cout << "total global memory: " << cuda_dev_properties.totalGlobalMem << "\n";
        cout << "memory bus width: " << cuda_dev_properties.memoryBusWidth << "\n";
        cout << "clock rate: " << cuda_dev_properties.clockRate << "\n";
        cout << "multiprocessor count: " << cuda_dev_properties.multiProcessorCount << "\n";
        cout << "max threads per multiprocessor: " << cuda_dev_properties.maxThreadsPerMultiProcessor << "\n";
        cout << "registers per multiprocessor: " << cuda_dev_properties.regsPerMultiprocessor << "\n";
        cout << "warp size: " << cuda_dev_properties.warpSize << "\n";
        cout << "registers per block: " << cuda_dev_properties.regsPerBlock << "\n";
        cout << "shared memory per block: " << cuda_dev_properties.sharedMemPerBlock << "\n";
        cout << "max dimension size of a grid (x, y, z): " 
            << cuda_dev_properties.maxGridSize[0] << ", " 
            << cuda_dev_properties.maxGridSize[1] << ", " 
            << cuda_dev_properties.maxGridSize[2] << "\n";
        cout << "max dimension size of a thread block (x, y, z): " 
            << cuda_dev_properties.maxThreadsDim[0] << ", " 
            << cuda_dev_properties.maxThreadsDim[1] << ", " 
            << cuda_dev_properties.maxThreadsDim[2] << "\n";
        cout << "concurrent kernels: " << cuda_dev_properties.concurrentKernels << "\n";
        cout << "local L1 cache supported: " 
            << (cuda_dev_properties.localL1CacheSupported ? "yes" : "no") << "\n";
        if (cuda_dev_properties.totalGlobalMem < total_data_bytes) {
            cout << "** CUDA device insufficient global memory; need " << total_data_bytes << " bytes for data alone; " << cuda_dev_properties.totalGlobalMem << " on device\n";
        }
    }
}

// execute kernels on the specific device
//  - profiles with CUPTI
//  - checks the data output to ensure every point was properly calculated
void run_calculations(size_t num_points, size_t block_size, size_t step_size, int dev_num) {
    CUDA_ERR_SAFE( hipSetDevice(dev_num) );
    size_t num_blocks = (num_points + block_size - 1) / block_size;
    // some extra blocks for when step size does not divide evenly into block size, to
    //   ensure the last data points get calculated
    num_blocks += ((num_blocks % step_size) > 0) ? step_size - (num_blocks % step_size) : 0;
    cout << "Allocating and initializing " << num_points << " doubles\n";
    double *host_data = new double[num_points];
    size_t num_point_bytes = num_points * sizeof(double);
    // meaningless but verifiable data: set each element to one-tenth its position in the array
    for (size_t index = 0; index < num_points; ++index) {
        host_data[index] = static_cast<double>(index) / 10.0;
    }
    // data_dump(host_data, num_points); // TODO DEBUG

    // run warm-up before activating CUPTI
    warm_up<<<num_blocks, block_size>>>();

    double *device_data;
    CUDA_ERR_SAFE( hipMalloc(&device_data, num_point_bytes) );
    CUDA_ERR_SAFE( hipMemcpy(device_data, host_data, num_point_bytes, hipMemcpyHostToDevice) );

    CUPTI_ERR_SAFE( cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL) );
    CUPTI_ERR_SAFE( cuptiActivityEnable(CUPTI_ACTIVITY_KIND_SHARED_ACCESS) );
    CUPTI_ERR_SAFE( cuptiActivityRegisterCallbacks(cuptiBufferRequested, cuptiBufferCompleted) );

    CUDA_ERR_SAFE( hipGetLastError() );

    // Compute!
    cout << "Running " << num_blocks << " blocks of size " << block_size << " with step size " << step_size << "\n";
    calculate_with_step_shared_mem<<<num_blocks, block_size, block_size * sizeof(double) * step_size>>>(device_data, step_size, num_points);

    CUDA_ERR_SAFE( hipDeviceSynchronize() );
    CUDA_ERR_SAFE( hipGetLastError() );

    CUPTI_ERR_SAFE( cuptiActivityFlushAll(0) );

    CUDA_ERR_SAFE( hipMemcpy(host_data, device_data, num_point_bytes, hipMemcpyDeviceToHost) );

    // check computation correctness; verifies indexes were properly calculated in the kernel(s)
    // data_dump(host_data, num_points); // TODO DEBUG
    for (size_t index = 0; index < num_points; ++index) {
        const double original_value = static_cast<double>(index) / 10.0;
        const double new_value = host_data[index];
        const double expected_value = (original_value * original_value * 10.0);
        if ( !doubles_within_tolerance(new_value, expected_value) ) {
            cerr << "** DATA ERROR **: Result data incorrect at index " << index
                 << "; expected " << (original_value * original_value * 10.0)
                 << " and encountered " << host_data[index] << "\n";
            // only need to report a single incorrect value
            break;
        }
    }

    CUDA_ERR_SAFE( hipFree(device_data) );
    delete[] host_data;
    CUDA_ERR_SAFE( hipDeviceReset() );
}
